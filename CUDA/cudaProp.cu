#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>

// 输出CUDA内置的相关信息


int main(){
    hipDeviceProp_t prop;
    int count = 0;
    hipGetDeviceCount(&count);
    for (int i = 0; i < count; i++){
        hipGetDeviceProperties(&prop, i);
    }
    std::cout << "Device name: " << prop.name << std::endl;
    std::cout << "Kernel version: code_" << prop.major  << prop.minor << std::endl;
    std::cout << "Total Global Mem: " << prop.totalGlobalMem << std::endl;
    std::cout << "Shared Mem per block: " << prop.sharedMemPerBlock << std::endl;
    std::cout << "Warp size: " << prop.warpSize << std::endl;
    std::cout << "Max thread per block: " << prop.maxThreadsPerBlock << std::endl;
    std::cout << "Max thread dim: " << prop.maxThreadsDim[0] << ", " << prop.maxThreadsDim[1] <<
             ", " << prop.maxThreadsDim[2] << std::endl;
    std::cout << "Max Grid size: " << prop.maxGridSize[0] << ", " << prop.maxGridSize[1] <<
             ", " << prop.maxGridSize[2] << std::endl;
    return 0;
}